#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include "lib/cycletimer.h"
#include "lib/etc.h"
#include "lib/ppm.h"

#ifdef DEBUG
#define CUDA_CHECK cudaCheck(hipPeekAtLastError(), __FILE__, __LINE__)
static inline void cudaCheck(hipError_t code, const char *file, int line) {
    if (code == hipSuccess) {
        return;
    }
    fprintf(stderr, "%s:%d: %s\n", file, line, hipGetErrorString(code));
    exit(EXIT_FAILURE);
}
#else
#define CUDA_CHECK ((void) 0)
#endif

static inline __host__ __device__ int div_ceil(int n, int d) {
    return (n + (d - 1)) / d;
}


static inline __device__ int cudaGetBucketIdx(int r, int g, int b) {
    return r * BUCKETS * BUCKETS + g * BUCKETS + b;
}

__global__ void getColorDist(
    int width,
    int height,
    int *color_counts,
    PPMPixel *imgData,
    int ltWall,
    int rtWall,
    int tpWall
) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= height || j >= width) {
        return;
    }
    if (j >= ltWall && j < rtWall && i > tpWall) {
        return;
    }

    PPMPixel pt = imgData[i * width + j];
    int bucketIdx = cudaGetBucketIdx(
                    pt.red / BUCKET_SIZE,
                    pt.green / BUCKET_SIZE,
                    pt.blue / BUCKET_SIZE);

    atomicAdd(&color_counts[bucketIdx], 1);
}

__global__ void initMask(
    int width,
    int height,
    char *oldMask,
    int *color_counts,
    PPMPixel *imgData,
    int bcThresh
) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= height || j >= width) {
        return;
    }

    PPMPixel pt = imgData[i * width + j];
    unsigned char r = pt.red / BUCKET_SIZE;
    unsigned char g = pt.green / BUCKET_SIZE;
    unsigned char b = pt.blue / BUCKET_SIZE;
    if (color_counts[cudaGetBucketIdx(r, g, b)] < bcThresh) {
        oldMask[i * width + j] = 1;
    }
}

__global__ void buildMask(
    int width,
    int height,
    char *oldMask,
    char *mask
) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < 2 || j < 2 || i >= height - 2 || j >= width - 2) {
        return;
    }

    __syncthreads();

    // Clean up mask
    char thisPx = oldMask[i * width + j];
    if (thisPx == 0) {
        int borderSum =
            oldMask[(i - 1) * width + j] +
            oldMask[i * width + j - 1] +
            oldMask[(i + 1) * width + j] +
            oldMask[i * width + j + 1] +
            oldMask[(i - 2) * width + j] +
            oldMask[i * width + j - 2] +
            oldMask[(i + 2) * width + j] +
            oldMask[i * width + j + 2];
        if (borderSum >= 2) {
            mask[i * width + j] = 1;
        }
    }
}

__global__ void blur(
    int width,
    int height,
    PPMPixel *imgData,
    float *blurKernel,
    PPMPixel *blurData,
    char *mask
) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int sqIdx = threadIdx.y * SQ_DIM + threadIdx.x;

    // Load Kernel into shared mem
    __shared__ float sharedBlurKernel[FILTER_SIZE * FILTER_SIZE];
    int blurKernelCopyLen = div_ceil(FILTER_SIZE * FILTER_SIZE,
                                    SQ_DIM * SQ_DIM);
    int index;
    for (int ind = 0; ind < blurKernelCopyLen; ind++) {
        index = ind + sqIdx * blurKernelCopyLen;
        if (index >= FILTER_SIZE * FILTER_SIZE) {
            continue;
        }
        sharedBlurKernel[index] = blurKernel[index];
    }

    // Load image into shared memory
    __shared__ PPMPixel sharedImgData[SHARED_IMG_DATA_DIM*SHARED_IMG_DATA_DIM];
    int imgDataCopyLen = div_ceil(SHARED_IMG_DATA_DIM * SHARED_IMG_DATA_DIM,
                                  SQ_DIM * SQ_DIM);

    int imgIndex;
    int rowOffset = blockIdx.y * SQ_DIM - (FILTER_SIZE / 2);
    int colOffset = blockIdx.x * SQ_DIM - (FILTER_SIZE / 2);

    for (int ind = 0; ind < imgDataCopyLen; ind++) {

        index = ind + sqIdx * imgDataCopyLen;
        int imgRow = rowOffset + (index / SHARED_IMG_DATA_DIM);
        int imgCol = colOffset + (index % SHARED_IMG_DATA_DIM);

        imgIndex = imgRow * width + imgCol;
        if (imgRow < 0 || imgCol < 0) {
            continue;
        }
        else if (index < 0 || index >= SHARED_IMG_DATA_DIM * SHARED_IMG_DATA_DIM) {
            continue;
        }
        else if (imgIndex < 0 || imgIndex >= width * height) {
            continue;
        }
        sharedImgData[index] = imgData[imgIndex];
    }

    __syncthreads();

    if (row < 0 || row >= height || col < 0 || col >= width) {
        return;
    }
    if (mask[row * width + col] == 1) {
        return;
    }

    float count = 0;
    int i_k, j_k;
    float red = 0;
    float green = 0;
    float blue = 0;
    for (i_k = 0; i_k < FILTER_SIZE; i_k++){
        for (j_k = 0; j_k < FILTER_SIZE; j_k++){
            float weight = sharedBlurKernel[i_k*FILTER_SIZE + j_k];
            int i = row - (FILTER_SIZE / 2) + i_k;
            int j = col - (FILTER_SIZE / 2) + j_k;

            if (i < 0 || i >= height || j < 0 || j >= width) {
                continue;
            } else if (mask[i * width + j] == 1) {
                continue;
            } else if (i - rowOffset < 0 || i - rowOffset >= SHARED_IMG_DATA_DIM
                       || j - colOffset < 0
                       || j - colOffset >= SHARED_IMG_DATA_DIM) {
                continue;
            }
            PPMPixel pt = sharedImgData[SHARED_IMG_DATA_DIM * (i - rowOffset) + j - colOffset];
            //PPMPixel pt = imgData[width * (i) + j];
            red += weight * (pt.red);
            green += weight * (pt.green);
            blue += weight * (pt.blue);
            count += weight;
        }
    }

    if (count != 0) {
        blurData[row * width + col].red = (unsigned char)(red / count);
        blurData[row * width + col].green = (unsigned char)(green / count);
        blurData[row * width + col].blue = (unsigned char)(blue / count);
    }
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("usage: %s <infile> <outfile>\n", argv[0]);
        exit(EXIT_FAILURE);
    }
    char *infile = argv[1];
    char *outfile = argv[2];

    double start;

    printf("begin\n");
    start = currentSeconds();

    PPMImage *img = readPPM(infile);
    if (img == NULL) {
        exit(EXIT_FAILURE);
    }

    printf("load image: %lf\n", currentSeconds() - start);
    start = currentSeconds();

    int *color_counts =
        (int *)calloc(BUCKETS * BUCKETS * BUCKETS, sizeof(int));
    char *oldMask =
        (char *)calloc(img->width * img->height, sizeof(char));
    char *mask =
        (char *)calloc(img->width * img->height, sizeof(char));
    float *blurKernel =
        (float *)calloc(FILTER_SIZE * FILTER_SIZE, sizeof(float));
    PPMPixel *blurData =
        (PPMPixel *)calloc(img->width * img->height, sizeof(PPMPixel));

    if (
        color_counts == NULL ||
        oldMask == NULL ||
        mask == NULL ||
        blurKernel == NULL ||
        blurData == NULL
    ) {
        exit(EXIT_FAILURE);
    }

    PPMPixel *cudaImgData;
    hipMalloc(&cudaImgData, img->width * img->height * sizeof(PPMPixel));
    hipMemcpy(cudaImgData, img->data,
        img->width * img->height * sizeof(PPMPixel),
        hipMemcpyHostToDevice);

    PPMPixel *cudaBlurData;
    hipMalloc(&cudaBlurData, img->width * img->height * sizeof(PPMPixel));
    hipMemcpy(cudaBlurData, img->data,
        img->width * img->height * sizeof(PPMPixel),
        hipMemcpyHostToDevice);

    // Even bokeh circle blur
    for (int i = 0; i < FILTER_SIZE; i++) {
        for (int j = 0; j < FILTER_SIZE; j++) {
            int x = (FILTER_SIZE/2) - j;
            int y = (FILTER_SIZE/2) - i;
            if (x * x + y * y < (FILTER_SIZE/2) * (FILTER_SIZE/2)) {
                blurKernel[i * FILTER_SIZE + j] = 1.0;
            }
        }
    }
    float *cudaBlurKernel;
    hipMalloc(&cudaBlurKernel, FILTER_SIZE * FILTER_SIZE * sizeof(float));
    hipMemcpy(cudaBlurKernel, blurKernel,
        FILTER_SIZE * FILTER_SIZE * sizeof(float),
        hipMemcpyHostToDevice);

    int *cudaColorCounts;
    hipMalloc(&cudaColorCounts, BUCKETS * BUCKETS * BUCKETS * sizeof(int));

    char *cudaOldMask;
    hipMalloc(&cudaOldMask, img->width * img->height * sizeof(char));

    char *cudaMask;
    hipMalloc(&cudaMask, img->width * img->height * sizeof(char));

    printf("malloc and cudamalloc and memcpy: %lf\n", currentSeconds() - start);
    start = currentSeconds();
    // Get Walls
    int ltWall = img->width / LTRTWALLDENOM;
    int rtWall = (img->width * (LTRTWALLDENOM - 1)) / LTRTWALLDENOM;
    int tpWall = img->height / TPWALLDENOM;

    // Get color distribution

    hipMemcpy(cudaColorCounts, color_counts,
        BUCKETS * BUCKETS * BUCKETS * sizeof(int),
        hipMemcpyHostToDevice);

    // Dims for every pixel
    dim3 threadsPerBlock(SQ_DIM, SQ_DIM);
    dim3 blocks(div_ceil(img->width, SQ_DIM), div_ceil(img->height, SQ_DIM));

    getColorDist<<<blocks, threadsPerBlock>>>(
        img->width,
        img->height,
        cudaColorCounts,
        cudaImgData,
        ltWall,
        rtWall,
        tpWall
    );
    CUDA_CHECK;

    printf("get color_counts: %lf\n", currentSeconds() - start);
    start = currentSeconds();

    int totalBCPix =
        ltWall * img->height +
        (img->width - rtWall) * img->height +
        tpWall * img->width;

    int bcThresh = BCTHRESH_DECIMAL * totalBCPix;

    hipMemcpy(cudaOldMask, oldMask,
        img->width * img->height * sizeof(char),
        hipMemcpyHostToDevice);

    initMask<<<blocks, threadsPerBlock>>>(
        img->width,
        img->height,
        cudaOldMask,
        cudaColorCounts,
        cudaImgData,
        bcThresh
    );

    printf("get oldMask: %lf\n", currentSeconds() - start);
    start = currentSeconds();

    hipMemcpy(cudaMask, cudaOldMask,
        img->width * img->height * sizeof(char),
        hipMemcpyDeviceToDevice);

    buildMask<<<blocks, threadsPerBlock>>>(
        img->width,
        img->height,
        cudaOldMask,
        cudaMask
    );

    printf("get mask: %lf\n", currentSeconds() - start);
    start = currentSeconds();

    // Blur
    printf("finished mask, starting blur\n");

    blur<<<blocks, threadsPerBlock>>>(
        img->width,
        img->height,
        cudaImgData,
        cudaBlurKernel,
        cudaBlurData,
        cudaMask
    );

    hipDeviceSynchronize();
    hipMemcpy(
        blurData,
        cudaBlurData,
        img->width * img->height * sizeof(PPMPixel),
        hipMemcpyDeviceToHost
    );

    printf("get blurData: %lf\n", currentSeconds() - start);
    start = currentSeconds();

    PPMPixel *oldData = img->data;
    img->data = blurData;

    errno = 0;
    writePPM(outfile, img);
    if (errno != 0) {
        exit(EXIT_FAILURE);
    }
    printf("write image: %lf\n", currentSeconds() - start);

    free(oldData);
    free(color_counts);
    free(blurKernel);
    free(img);
    free(img->data);
    hipFree(cudaImgData);
    hipFree(cudaBlurKernel);
    hipFree(cudaMask);
    hipFree(cudaOldMask);
    return 0;
}
