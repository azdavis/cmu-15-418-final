
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define RGB_COMPONENT_COLOR 255
#define LTRTWALLDENOM 7
#define TPWALLDENOM 8
#define BUCKET_SIZE 32
#define COLORS 256
#define BCTHRESH_DECIMAL 0.005
#define FILTER_SIZE 50
#define BUCKETS (COLORS / BUCKET_SIZE)

typedef struct {
     int xmin, xmax, ymin, ymax;
} range;

// PPM reading writing code guided from:
// https://stackoverflow.com/questions/2693631/read-ppm-file-and-store-it-in-an-array-coded-with-c

typedef struct {
     unsigned char red,green,blue;
} PPMPixel;

typedef struct {
     int width, height;
     PPMPixel *data;
} PPMImage;

static PPMImage *readPPM(const char *filename)
{
    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    //open PPM file for reading
    fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    //read image format
    if (!fgets(buff, sizeof(buff), fp)) {
      perror(filename);
      exit(1);
    }

    //check the image format
    if (buff[0] != 'P' || buff[1] != '6') {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    //alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //check for comments
    c = getc(fp);
    while (c == '#') {
    while (getc(fp) != '\n') ;
         c = getc(fp);
    }

    ungetc(c, fp);
    //read image size information
    if (fscanf(fp, "%d %d", &img->width, &img->height) != 2) {
         fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
         exit(1);
    }

    //read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
         fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
         exit(1);
    }

    //check rgb component depth
    if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
         fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
         exit(1);
    }

    while (fgetc(fp) != '\n') ;
    //memory allocation for pixel data
    img->data = (PPMPixel*)malloc(img->width * img->height * sizeof(PPMPixel));

    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //read pixel data from file
    if (fread(img->data, 3 * img->width, img->height, fp) != img->height) {
         fprintf(stderr, "Error loading image '%s'\n", filename);
         exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    //open file for output
    fp = fopen(filename, "wb");
    if (!fp) {
         fprintf(stderr, "Unable to open file '%s'\n", filename);
         exit(1);
    }

    //write the header file
    //image format
    fprintf(fp, "P6\n");

    //image size
    fprintf(fp, "%d %d\n",img->width,img->height);

    // rgb component depth
    fprintf(fp, "%d\n",RGB_COMPONENT_COLOR);

    // pixel data
    fwrite(img->data, 3 * img->width, img->height, fp);
    fclose(fp);
}

static PPMPixel* getPixel(int x, int y, PPMImage *img)
{
    return &(img->data[x + y * img->width]);
}

static void setPixel(int x, int y, PPMImage *img,
                     unsigned char R, unsigned char G, unsigned char B)
{
    img->data[x + y * img->width].red = R;
    img->data[x + y * img->width].green = G;
    img->data[x + y * img->width].blue = B;
}

static int getBucketIdx(int r, int g, int b)
{
    return r * BUCKETS * BUCKETS + g * BUCKETS + b;
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("usage: %s <infile> <outfile>\n", argv[0]);
        return 0;
    }
    char *infile = argv[1];
    char *outfile = argv[2];
    PPMImage *img = readPPM(infile);

    // Get Walls
    int ltWall = img->width / LTRTWALLDENOM;
    int rtWall = (img->width * (LTRTWALLDENOM - 1)) / LTRTWALLDENOM;
    int tpWall = img->height / TPWALLDENOM;

    // Get color distribution
    //int buckets = COLORS / BUCKET_SIZE;
    int *color_counts = (int*) malloc(BUCKETS * BUCKETS * BUCKETS * sizeof(int));
    if (color_counts == NULL)
        exit(1);

    range rs[3];
    rs[0].xmin = 0; rs[0].xmax = ltWall;
    rs[0].ymin = 0; rs[0].ymax = img->height;
    rs[1].xmin = rtWall; rs[1].xmax = img->width;
    rs[1].ymin = 0; rs[1].ymax = img->height;
    rs[2].xmin = 0; rs[2].xmax = img->width;
    rs[2].ymin = 0; rs[2].ymax = tpWall;

    int i, j, ri;
    for (ri = 0; ri < 3; ri++) {
        range r = rs[ri];
        for (i = r.ymin; i < r.ymax; i++) {
            for (j = r.xmin; j < r.xmax; j++) {
                PPMPixel *pt = getPixel(j, i, img);
                color_counts[getBucketIdx(pt->red / BUCKET_SIZE,
                                          pt->green / BUCKET_SIZE,
                                          pt->blue / BUCKET_SIZE)] += 1;

            }
        }
    }

    int totalBCPix = (ltWall * img->height +
                     (img->width - rtWall) * img->height +
                      tpWall * img->width);
    int bcThresh = BCTHRESH_DECIMAL * totalBCPix;

    char *oldMask = (char*) calloc(img->width * img->height, sizeof(char));
    if (oldMask == NULL)
        exit(1);

    for (i = 0; i < img->height; i++) {
        for (j = 0; j < img->width; j++) {
            PPMPixel *pt = getPixel(j, i, img);
            unsigned char r = pt->red / BUCKET_SIZE;
            unsigned char g = pt->green / BUCKET_SIZE;
            unsigned char b = pt->blue / BUCKET_SIZE;
            if (color_counts[getBucketIdx(r, g, b)] < bcThresh) {
                oldMask[i * img->width + j] = 1;
            }
        }
    }

    char *mask = (char*) calloc(img->width * img->height, sizeof(char));
    if (mask == NULL)
        exit(1);
    memcpy(mask, oldMask, img->width * img->height * sizeof(char));

    // Clean up mask
    for (i = 2; i < img->height-2; i++) {
        for (j = 2; j < img->width-2; j++) {
            char thisPx = oldMask[i * img->width + j];
            if (thisPx == 0) {
                int borderSum = (oldMask[(i-1) * img->width + j] +
                                 oldMask[i * img->width + j-1] +
                                 oldMask[(i+1) * img->width + j] +
                                 oldMask[i * img->width + j+1] +
                                 oldMask[(i-2) * img->width + j] +
                                 oldMask[i * img->width + j-2] +
                                 oldMask[(i+2) * img->width + j] +
                                 oldMask[i * img->width + j+2]);
                if (borderSum >= 2) {
                   mask[i * img->width + j] = 1;
                }
            }
        }
    }

    // Blur
    printf("finished mask, starting blur\n");
    float *blurKernel = (float*) malloc(FILTER_SIZE * FILTER_SIZE * sizeof(float));
    if (blurKernel == NULL)
        exit(1);
    // Even box blur
    for (i = 0; i < FILTER_SIZE; i++) {
        for (j = 0; j < FILTER_SIZE; j++) {
            blurKernel[i * FILTER_SIZE + j] = 1.0;
        }
    }

    PPMPixel *blurData = (PPMPixel*) calloc(img->width * img->height, sizeof(PPMPixel));
    if (blurData == NULL)
        exit(1);

    int width = img->width;
    int height = img->height;
    int row, col;
    for (row = 0; row < height; row++) {
        if (row % 10 == 0) {
            printf("finished row %d\n", row);
        }
        for (col = 0; col < width; col++) {
            float count = 0;
            int i_k, j_k;
            float red = 0;
            float green = 0;
            float blue = 0;
            for (i_k = 0; i_k < FILTER_SIZE; i_k++){
                for (j_k = 0; j_k < FILTER_SIZE; j_k++){
                    float weight = blurKernel[i_k*FILTER_SIZE + j_k];
                    int i = row - (FILTER_SIZE / 2) + i_k;
                    int j = col - (FILTER_SIZE / 2) + j_k;

                    if (i < 0 || i >= height || j < 0 || j >= width) {
                        continue;
                    }
                    else if (mask[i * width + j] == 1) {
                        continue;
                    }
                    PPMPixel *pt = getPixel(j, i, img);
                    red += weight * (pt->red);
                    green += weight * (pt->green);
                    blue += weight * (pt->blue);
                    count += weight;
                }
            }
            if (count == 0) {
                continue;
            }

            blurData[row*width + col].red = (unsigned char) (red / count);
            blurData[row*width + col].green = (unsigned char) (green / count);
            blurData[row*width + col].blue = (unsigned char) (blue / count);
        }
    }
    // Put filter on mask
    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            if (mask[i*width + j] == 1) {
                PPMPixel *pt = getPixel(j, i, img);
                blurData[i*width + j].red = pt->red;
                blurData[i*width + j].green = pt->green;
                blurData[i*width + j].blue = pt->blue;
            }
        }
    }

    PPMPixel *oldData = img->data;
    img->data = blurData;

    writePPM(outfile, img);

    free(oldData);
    free(color_counts);
    free(blurKernel);
    free(img);
    free(img->data);
    return 0;
}
