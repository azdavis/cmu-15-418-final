
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <string>

#define RGB_COMPONENT_COLOR 255
#define LTRTWALLDENOM 7
#define TPWALLDENOM 8
#define BUCKET_SIZE 32
#define COLORS 256
#define BCTHRESH_DECIMAL 0.005
#define FILTER_SIZE 50
#define BUCKETS (COLORS / BUCKET_SIZE)

#if 1
#define CUDA_CHECK cudaCheck(hipPeekAtLastError(), __FILE__, __LINE__)
static inline void cudaCheck(hipError_t code, const char *file, int line) {
    if (code == hipSuccess) {
        return;
    }
    fprintf(stderr, "%s:%d: %s\n", file, line, hipGetErrorString(code));
    exit(EXIT_FAILURE);
}
#else
#define CUDA_CHECK ((void) 0)
#endif


typedef struct {
     int xmin, xmax, ymin, ymax;
} range;

// PPM reading writing code guided from:
// https://stackoverflow.com/questions/2693631/read-ppm-file-and-store-it-in-an-array-coded-with-c

typedef struct {
     unsigned char red,green,blue;
} PPMPixel;

typedef struct {
     int width, height;
     PPMPixel *data;
} PPMImage;

static PPMImage *readPPM(const char *filename)
{
    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    //open PPM file for reading
    fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    //read image format
    if (!fgets(buff, sizeof(buff), fp)) {
      perror(filename);
      exit(1);
    }

    //check the image format
    if (buff[0] != 'P' || buff[1] != '6') {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    //alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //check for comments
    c = getc(fp);
    while (c == '#') {
    while (getc(fp) != '\n') ;
         c = getc(fp);
    }

    ungetc(c, fp);
    //read image size information
    if (fscanf(fp, "%d %d", &img->width, &img->height) != 2) {
         fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
         exit(1);
    }

    //read rgb component
    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
         fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
         exit(1);
    }

    //check rgb component depth
    if (rgb_comp_color!= RGB_COMPONENT_COLOR) {
         fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
         exit(1);
    }

    while (fgetc(fp) != '\n') ;
    //memory allocation for pixel data
    img->data = (PPMPixel*)malloc(img->width * img->height * sizeof(PPMPixel));

    if (!img) {
         fprintf(stderr, "Unable to allocate memory\n");
         exit(1);
    }

    //read pixel data from file
    if (fread(img->data, 3 * img->width, img->height, fp) != img->height) {
         fprintf(stderr, "Error loading image '%s'\n", filename);
         exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    //open file for output
    fp = fopen(filename, "wb");
    if (!fp) {
         fprintf(stderr, "Unable to open file '%s'\n", filename);
         exit(1);
    }

    //write the header file
    //image format
    fprintf(fp, "P6\n");

    //image size
    fprintf(fp, "%d %d\n",img->width,img->height);

    // rgb component depth
    fprintf(fp, "%d\n",RGB_COMPONENT_COLOR);

    // pixel data
    fwrite(img->data, 3 * img->width, img->height, fp);
    fclose(fp);
}

static PPMPixel* getPixel(int x, int y, PPMImage *img)
{
    return &(img->data[x + y * img->width]);
}

static void setPixel(int x, int y, PPMImage *img,
                     unsigned char R, unsigned char G, unsigned char B)
{
    img->data[x + y * img->width].red = R;
    img->data[x + y * img->width].green = G;
    img->data[x + y * img->width].blue = B;
}

static int getBucketIdx(int r, int g, int b)
{
    return r * BUCKETS * BUCKETS + g * BUCKETS + b;
}

__host__ int main(int argc, char **argv) {
    if (argc != 3) {
        printf("usage: %s <infile> <outfile>\n", argv[0]);
        return 0;
    }
    char *infile = argv[1];
    char *outfile = argv[2];

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf(
            "   Global mem: %.0f MB\n",
            static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024)
        );
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }

    PPMImage *img = readPPM(infile);

    int *color_counts = (int*) malloc(BUCKETS * BUCKETS * BUCKETS * sizeof(int));
    if (color_counts == NULL)
        exit(1);
    char *oldMask = (char*) calloc(img->width * img->height, sizeof(char));
    if (oldMask == NULL)
        exit(1);
    char *mask = (char*) calloc(img->width * img->height, sizeof(char));
    if (mask == NULL)
        exit(1);
    float *blurKernel = (float*) malloc(FILTER_SIZE * FILTER_SIZE * sizeof(float));
    if (blurKernel == NULL)
        exit(1);
    PPMPixel *blurData = (PPMPixel*) calloc(img->width * img->height, sizeof(PPMPixel));
    if (blurData == NULL)
        exit(1);

    PPMPixel *cudaImgData;
    hipMalloc(&cudaImgData, img->width * img->height * sizeof(PPMPixel));
    hipMemcpy(cudaImgData,
               img->data,
               img->width * img->height * sizeof(PPMPixel),
               hipMemcpyHostToDevice
    );

    float *cudaBlurKernel;
    hipMalloc(&cudaBlurKernel, FILTER_SIZE * FILTER_SIZE * sizeof(float));
    hipMemcpy(cudaBlurKernel,
               blurKernel,
               FILTER_SIZE * FILTER_SIZE * sizeof(float),
               hipMemcpyHostToDevice
    );

    // Get Walls
    int ltWall = img->width / LTRTWALLDENOM;
    int rtWall = (img->width * (LTRTWALLDENOM - 1)) / LTRTWALLDENOM;
    int tpWall = img->height / TPWALLDENOM;

    // Get color distribution
    range rs[3];
    rs[0].xmin = 0; rs[0].xmax = ltWall;
    rs[0].ymin = 0; rs[0].ymax = img->height;
    rs[1].xmin = rtWall; rs[1].xmax = img->width;
    rs[1].ymin = 0; rs[1].ymax = img->height;
    rs[2].xmin = 0; rs[2].xmax = img->width;
    rs[2].ymin = 0; rs[2].ymax = tpWall;

    int i, j, ri;
    for (ri = 0; ri < 3; ri++) {
        range r = rs[ri];
        for (i = r.ymin; i < r.ymax; i++) {
            for (j = r.xmin; j < r.xmax; j++) {
                PPMPixel *pt = getPixel(j, i, img);
                color_counts[getBucketIdx(pt->red / BUCKET_SIZE,
                                          pt->green / BUCKET_SIZE,
                                          pt->blue / BUCKET_SIZE)] += 1;

            }
        }
    }

    int totalBCPix = (ltWall * img->height +
                     (img->width - rtWall) * img->height +
                      tpWall * img->width);
    int bcThresh = BCTHRESH_DECIMAL * totalBCPix;

    for (i = 0; i < img->height; i++) {
        for (j = 0; j < img->width; j++) {
            PPMPixel *pt = getPixel(j, i, img);
            unsigned char r = pt->red / BUCKET_SIZE;
            unsigned char g = pt->green / BUCKET_SIZE;
            unsigned char b = pt->blue / BUCKET_SIZE;
            if (color_counts[getBucketIdx(r, g, b)] < bcThresh) {
                oldMask[i * img->width + j] = 1;
            }
        }
    }

    memcpy(mask, oldMask, img->width * img->height * sizeof(char));

    // Clean up mask
    for (i = 2; i < img->height-2; i++) {
        for (j = 2; j < img->width-2; j++) {
            char thisPx = oldMask[i * img->width + j];
            if (thisPx == 0) {
                int borderSum = (oldMask[(i-1) * img->width + j] +
                                 oldMask[i * img->width + j-1] +
                                 oldMask[(i+1) * img->width + j] +
                                 oldMask[i * img->width + j+1] +
                                 oldMask[(i-2) * img->width + j] +
                                 oldMask[i * img->width + j-2] +
                                 oldMask[(i+2) * img->width + j] +
                                 oldMask[i * img->width + j+2]);
                if (borderSum >= 2) {
                   mask[i * img->width + j] = 1;
                }
            }
        }
    }

    // Blur
    printf("finished mask, starting blur\n");
    // Even box blur
    for (i = 0; i < FILTER_SIZE; i++) {
        for (j = 0; j < FILTER_SIZE; j++) {
            blurKernel[i * FILTER_SIZE + j] = 1.0;
        }
    }

    int width = img->width;
    int height = img->height;
    int row, col;
    for (row = 0; row < height; row++) {
        if (row % 100 == 0) {
            printf("finished row %d\n", row);
        }
        for (col = 0; col < width; col++) {
            float count = 0;
            int i_k, j_k;
            float red = 0;
            float green = 0;
            float blue = 0;
            for (i_k = 0; i_k < FILTER_SIZE; i_k++){
                for (j_k = 0; j_k < FILTER_SIZE; j_k++){
                    float weight = blurKernel[i_k*FILTER_SIZE + j_k];
                    int i = row - (FILTER_SIZE / 2) + i_k;
                    int j = col - (FILTER_SIZE / 2) + j_k;

                    if (i < 0 || i >= height || j < 0 || j >= width) {
                        continue;
                    }
                    else if (mask[i * width + j] == 1) {
                        continue;
                    }
                    PPMPixel *pt = getPixel(j, i, img);
                    red += weight * (pt->red);
                    green += weight * (pt->green);
                    blue += weight * (pt->blue);
                    count += weight;
                }
            }
            if (count == 0) {
                continue;
            }

            blurData[row*width + col].red = (unsigned char) (red / count);
            blurData[row*width + col].green = (unsigned char) (green / count);
            blurData[row*width + col].blue = (unsigned char) (blue / count);
        }
    }
    // Put filter on mask
    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            if (mask[i*width + j] == 1) {
                PPMPixel *pt = getPixel(j, i, img);
                blurData[i*width + j].red = pt->red;
                blurData[i*width + j].green = pt->green;
                blurData[i*width + j].blue = pt->blue;
            }
        }
    }

    PPMPixel *oldData = img->data;
    img->data = blurData;

    writePPM(outfile, img);

    free(oldData);
    free(color_counts);
    free(blurKernel);
    free(img);
    free(img->data);
    hipFree(cudaImgData);
    hipFree(cudaBlurKernel);
    return 0;
}
