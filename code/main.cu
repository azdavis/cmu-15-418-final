#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <string>
#include "ppm.h"

#define RGB_COMPONENT_COLOR 255
#define LTRTWALLDENOM 7
#define TPWALLDENOM 8
#define BUCKET_SIZE 32
#define COLORS 256
#define BCTHRESH_DECIMAL 0.005
#define FILTER_SIZE 50
#define BUCKETS (COLORS / BUCKET_SIZE)
#define SQ_DIM 32

#if 1
#define CUDA_CHECK cudaCheck(hipPeekAtLastError(), __FILE__, __LINE__)
static inline void cudaCheck(hipError_t code, const char *file, int line) {
    if (code == hipSuccess) {
        return;
    }
    fprintf(stderr, "%s:%d: %s\n", file, line, hipGetErrorString(code));
    exit(EXIT_FAILURE);
}
#else
#define CUDA_CHECK ((void)0)
#endif

typedef struct {
    int xmin, xmax, ymin, ymax;
} range;

static inline __host__ __device__ int div_ceil(int n, int d) {
    return (n + (d - 1)) / d;
}

static int getBucketIdx(int r, int g, int b) {
    return r * BUCKETS * BUCKETS + g * BUCKETS + b;
}

__global__ void blur(int width, int height, PPMPixel *imgData,
                                         float *blurKernel, PPMPixel *blurData, char *mask) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int sqIdx = threadIdx.y * SQ_DIM + threadIdx.x;

    // Load Kernel into shared mem
    __shared__ float sharedBlurKernel[FILTER_SIZE * FILTER_SIZE];
    int blurKernelCopyLen = div_ceil(FILTER_SIZE * FILTER_SIZE,
                                    SQ_DIM * SQ_DIM);
    for (int ind = 0; ind < blurKernelCopyLen; ind++) {
        int index = ind + sqIdx * blurKernelCopyLen;
        if (index >= FILTER_SIZE * FILTER_SIZE) {
            continue;
        }
        sharedBlurKernel[index] = blurKernel[index];
    }

    __syncthreads();

    if (row < 0 || row >= height || col < 0 || col >= width) {
        return;
    }

    float count = 0;
    int i_k, j_k;
    float red = 0;
    float green = 0;
    float blue = 0;
    for (i_k = 0; i_k < FILTER_SIZE; i_k++){
        for (j_k = 0; j_k < FILTER_SIZE; j_k++){
            float weight = sharedBlurKernel[i_k*FILTER_SIZE + j_k];
            int i = row - (FILTER_SIZE / 2) + i_k;
            int j = col - (FILTER_SIZE / 2) + j_k;

            if (i < 0 || i >= height || j < 0 || j >= width) {
                continue;
            } else if (mask[i * width + j] == 1) {
                continue;
            }
            PPMPixel pt = imgData[width * i + j];
            red += weight * (pt.red);
            green += weight * (pt.green);
            blue += weight * (pt.blue);
            count += weight;
        }
    }

    if (count != 0) {
        blurData[row * width + col].red = (unsigned char)(red / count);
        blurData[row * width + col].green = (unsigned char)(green / count);
        blurData[row * width + col].blue = (unsigned char)(blue / count);
    }
}

__host__ int main(int argc, char **argv) {
    if (argc != 3) {
        printf("usage: %s <infile> <outfile>\n", argv[0]);
        return 0;
    }
    char *infile = argv[1];
    char *outfile = argv[2];

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
                     static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }

    PPMImage *img = readPPM(infile);
    if (img == NULL) {
        exit(1);
    }

    int *color_counts = (int *)malloc(BUCKETS * BUCKETS * BUCKETS * sizeof(int));
    if (color_counts == NULL)
        exit(1);
    char *oldMask = (char *)calloc(img->width * img->height, sizeof(char));
    if (oldMask == NULL)
        exit(1);
    char *mask = (char *)calloc(img->width * img->height, sizeof(char));
    if (mask == NULL)
        exit(1);
    float *blurKernel =
            (float *)malloc(FILTER_SIZE * FILTER_SIZE * sizeof(float));
    if (blurKernel == NULL)
        exit(1);
    PPMPixel *blurData =
            (PPMPixel *)calloc(img->width * img->height, sizeof(PPMPixel));
    if (blurData == NULL)
        exit(1);

    PPMPixel *cudaImgData;
    hipMalloc(&cudaImgData, img->width * img->height * sizeof(PPMPixel));
    hipMemcpy(cudaImgData, img->data,
                         img->width * img->height * sizeof(PPMPixel),
                         hipMemcpyHostToDevice);

    PPMPixel *cudaBlurData;
    hipMalloc(&cudaBlurData, img->width * img->height * sizeof(PPMPixel));
    hipMemcpy(cudaBlurData, img->data,
                         img->width * img->height * sizeof(PPMPixel),
                         hipMemcpyHostToDevice);

    // Even box blur
    for (int i = 0; i < FILTER_SIZE; i++) {
        for (int j = 0; j < FILTER_SIZE; j++) {
            blurKernel[i * FILTER_SIZE + j] = 1.0;
        }
    }
    float *cudaBlurKernel;
    hipMalloc(&cudaBlurKernel, FILTER_SIZE * FILTER_SIZE * sizeof(float));
    hipMemcpy(cudaBlurKernel, blurKernel,
                         FILTER_SIZE * FILTER_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Get Walls
    int ltWall = img->width / LTRTWALLDENOM;
    int rtWall = (img->width * (LTRTWALLDENOM - 1)) / LTRTWALLDENOM;
    int tpWall = img->height / TPWALLDENOM;

    // Get color distribution
    range rs[3];
    rs[0].xmin = 0;
    rs[0].xmax = ltWall;
    rs[0].ymin = 0;
    rs[0].ymax = img->height;
    rs[1].xmin = rtWall;
    rs[1].xmax = img->width;
    rs[1].ymin = 0;
    rs[1].ymax = img->height;
    rs[2].xmin = 0;
    rs[2].xmax = img->width;
    rs[2].ymin = 0;
    rs[2].ymax = tpWall;

    int i, j, ri;
    for (ri = 0; ri < 3; ri++) {
        range r = rs[ri];
        for (i = r.ymin; i < r.ymax; i++) {
            for (j = r.xmin; j < r.xmax; j++) {
                PPMPixel *pt = getPixel(j, i, img);
                color_counts[getBucketIdx(pt->red / BUCKET_SIZE,
                                                                    pt->green / BUCKET_SIZE,
                                                                    pt->blue / BUCKET_SIZE)] += 1;
            }
        }
    }

    int totalBCPix = (ltWall * img->height + (img->width - rtWall) * img->height +
                                        tpWall * img->width);
    int bcThresh = BCTHRESH_DECIMAL * totalBCPix;

    for (i = 0; i < img->height; i++) {
        for (j = 0; j < img->width; j++) {
            PPMPixel *pt = getPixel(j, i, img);
            unsigned char r = pt->red / BUCKET_SIZE;
            unsigned char g = pt->green / BUCKET_SIZE;
            unsigned char b = pt->blue / BUCKET_SIZE;
            if (color_counts[getBucketIdx(r, g, b)] < bcThresh) {
                oldMask[i * img->width + j] = 1;
            }
        }
    }

    memcpy(mask, oldMask, img->width * img->height * sizeof(char));

    // Clean up mask
    for (i = 2; i < img->height - 2; i++) {
        for (j = 2; j < img->width - 2; j++) {
            char thisPx = oldMask[i * img->width + j];
            if (thisPx == 0) {
                int borderSum = (oldMask[(i - 1) * img->width + j] +
                                                 oldMask[i * img->width + j - 1] +
                                                 oldMask[(i + 1) * img->width + j] +
                                                 oldMask[i * img->width + j + 1] +
                                                 oldMask[(i - 2) * img->width + j] +
                                                 oldMask[i * img->width + j - 2] +
                                                 oldMask[(i + 2) * img->width + j] +
                                                 oldMask[i * img->width + j + 2]);
                if (borderSum >= 2) {
                    mask[i * img->width + j] = 1;
                }
            }
        }
    }

    // Blur
    printf("finished mask, starting blur\n");
    char *cudaMask;
    hipMalloc(&cudaMask, img->width * img->height * sizeof(char));
    hipMemcpy(cudaMask, mask, img->width * img->height * sizeof(char),
                         hipMemcpyHostToDevice);

    dim3 threadsPerBlock(SQ_DIM, SQ_DIM);
    dim3 blocks(div_ceil(img->width, SQ_DIM), div_ceil(img->height, SQ_DIM));

    CUDA_CHECK;
    blur<<<blocks, threadsPerBlock>>>(img->width, img->height, cudaImgData,
                                                                        cudaBlurKernel, cudaBlurData, cudaMask);
    hipDeviceSynchronize();
    hipMemcpy(blurData, cudaBlurData,
                         img->width * img->height * sizeof(PPMPixel),
                         hipMemcpyDeviceToHost);
    // Put filter on mask
    int height = img->height;
    int width = img->width;
    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            if (mask[i * width + j] == 1) {
                PPMPixel *pt = getPixel(j, i, img);
                blurData[i * width + j].red = pt->red;
                blurData[i * width + j].green = pt->green;
                blurData[i * width + j].blue = pt->blue;
            }
        }
    }

    PPMPixel *oldData = img->data;
    img->data = blurData;

    errno = 0;
    writePPM(outfile, img);
    if (errno != 0) {
        exit(1);
    }

    free(oldData);
    free(color_counts);
    free(blurKernel);
    free(img);
    free(img->data);
    hipFree(cudaImgData);
    hipFree(cudaBlurKernel);
    return 0;
}
