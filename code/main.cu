#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <string>
#include "lib/ppm.h"
#include "lib/cycletimer.h"

#define LTRTWALLDENOM 7
#define TPWALLDENOM 8
#define BUCKET_SIZE 32
#define COLORS 256
#define BCTHRESH_DECIMAL 0.005
#define FILTER_SIZE 50
#define BUCKETS (COLORS / BUCKET_SIZE)
#define SQ_DIM 32
#define SHARED_IMG_DATA_DIM (FILTER_SIZE + SQ_DIM)

#define CUDA_CHECK cudaCheck(hipPeekAtLastError(), __FILE__, __LINE__)
static inline void cudaCheck(hipError_t code, const char *file, int line) {
    if (code == hipSuccess) {
        return;
    }
    fprintf(stderr, "%s:%d: %s\n", file, line, hipGetErrorString(code));
    exit(EXIT_FAILURE);
}

typedef struct {
    int xmin, xmax, ymin, ymax;
} range;

static inline __host__ __device__ int div_ceil(int n, int d) {
    return (n + (d - 1)) / d;
}

static int getBucketIdx(int r, int g, int b) {
    return r * BUCKETS * BUCKETS + g * BUCKETS + b;
}

__global__ void blur(
    int width,
    int height,
    PPMPixel *imgData,
    float *blurKernel,
    PPMPixel *blurData,
    char *mask
) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int sqIdx = threadIdx.y * SQ_DIM + threadIdx.x;


    // Load Kernel into shared mem
    __shared__ float sharedBlurKernel[FILTER_SIZE * FILTER_SIZE];
    int blurKernelCopyLen = div_ceil(FILTER_SIZE * FILTER_SIZE,
                                    SQ_DIM * SQ_DIM);
    int index;
    for (int ind = 0; ind < blurKernelCopyLen; ind++) {
        index = ind + sqIdx * blurKernelCopyLen;
        if (index >= FILTER_SIZE * FILTER_SIZE) {
            continue;
        }
        sharedBlurKernel[index] = blurKernel[index];
    }

    // Load image into shared memory
    __shared__ PPMPixel sharedImgData[SHARED_IMG_DATA_DIM*SHARED_IMG_DATA_DIM];
    int imgDataCopyLen = div_ceil(SHARED_IMG_DATA_DIM * SHARED_IMG_DATA_DIM,
                                  SQ_DIM * SQ_DIM);

    int imgIndex;
    int rowOffset = blockIdx.x * SQ_DIM - (FILTER_SIZE / 2);
    int colOffset = blockIdx.y * SQ_DIM - (FILTER_SIZE / 2);

    int blockX = blockIdx.x;
    int blockY = blockIdx.y;


    if ((sqIdx == 0) && (blockX == 0) && (blockY == 0)) {
        printf("sqIdx %d blockX %d blockY %d rowOffset %d colOffset %d\n", sqIdx, blockIdx.x, blockIdx.y, rowOffset, colOffset);
        //printf("copy len %d , shared_img_dat_dim %d\n", imgDataCopyLen, SHARED_IMG_DATA_DIM);
    }
    for (int ind = 0; ind < imgDataCopyLen; ind++) {

        index = ind + sqIdx * imgDataCopyLen;
        int imgRow = rowOffset + (index / SHARED_IMG_DATA_DIM);
        int imgCol = colOffset + (index % SHARED_IMG_DATA_DIM);

        imgIndex = imgRow * width + imgCol;
        if (imgRow < 0 || imgCol < 0) {
            continue;
        }
        if (index < 0 || index >= SHARED_IMG_DATA_DIM * SHARED_IMG_DATA_DIM) {
            if (sqIdx == 0 && blockIdx.x == 2 && blockIdx.y == 2) {
                printf("ind %d writing from imgIndex %d to shared index %d\n", ind, imgIndex, index);
            }
            continue;
        }
        if (imgIndex < 0 || imgIndex >= width * height) {
            if (sqIdx == 0 && blockIdx.x == 2 && blockIdx.y == 2) {
                printf("ind %d writing from imgIndex %d to shared index %d\n", ind, imgIndex, index);
            }
            continue;
        }
        sharedImgData[index] = imgData[imgIndex];

        printf("ind %d writing from imgIndex %d to shared index %d blockX %d blockY %d innerRow %d innerCol %d imgRow %d imgCol %d\n", ind, imgIndex, index, blockIdx.x, blockIdx.y, index / SHARED_IMG_DATA_DIM, index % SHARED_IMG_DATA_DIM, imgRow, imgCol);
    }

    __syncthreads();

    if (row < 0 || row >= height || col < 0 || col >= width) {
        return;
    }

    float count = 0;
    int i_k, j_k;
    float red = 0;
    float green = 0;
    float blue = 0;
    for (i_k = 0; i_k < FILTER_SIZE; i_k++){
        for (j_k = 0; j_k < FILTER_SIZE; j_k++){
            float weight = sharedBlurKernel[i_k*FILTER_SIZE + j_k];
            int i = row - (FILTER_SIZE / 2) + i_k;
            int j = col - (FILTER_SIZE / 2) + j_k;

            if (i < 0 || i >= height || j < 0 || j >= width) {
                continue;
            } else if (mask[i * width + j] == 1) {
                continue;
            }
            PPMPixel pt = sharedImgData[SHARED_IMG_DATA_DIM * (i - rowOffset) + j - colOffset];
            //PPMPixel pt = imgData[width * (i) + j];
            red += weight * (pt.red);
            green += weight * (pt.green);
            blue += weight * (pt.blue);
            count += weight;
        }
    }

    if (count != 0) {
        blurData[row * width + col].red = (unsigned char)(red / count);
        blurData[row * width + col].green = (unsigned char)(green / count);
        blurData[row * width + col].blue = (unsigned char)(blue / count);
    }
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("usage: %s <infile> <outfile>\n", argv[0]);
        exit(EXIT_FAILURE);
    }
    char *infile = argv[1];
    char *outfile = argv[2];

    double start;

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for Portrait Mode\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
                     static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }

    printf("begin\n");
    start = currentSeconds();
    PPMImage *img = readPPM(infile);
    printf("load image: %lf\n", currentSeconds() - start);
    start = currentSeconds();

    int *color_counts = (int *)malloc(BUCKETS * BUCKETS * BUCKETS * sizeof(int));
    char *oldMask = (char *)calloc(img->width * img->height, sizeof(char));
    char *mask = (char *)calloc(img->width * img->height, sizeof(char));
    float *blurKernel =
            (float *)malloc(FILTER_SIZE * FILTER_SIZE * sizeof(float));
    PPMPixel *blurData =
            (PPMPixel *)calloc(img->width * img->height, sizeof(PPMPixel));
    if (
        img == NULL ||
        color_counts == NULL ||
        oldMask == NULL ||
        mask == NULL ||
        blurKernel == NULL ||
        blurData == NULL
    ) {
        exit(EXIT_FAILURE);
    }

    PPMPixel *cudaImgData;
    hipMalloc(&cudaImgData, img->width * img->height * sizeof(PPMPixel));
    hipMemcpy(cudaImgData, img->data,
        img->width * img->height * sizeof(PPMPixel),
        hipMemcpyHostToDevice);

    PPMPixel *cudaBlurData;
    hipMalloc(&cudaBlurData, img->width * img->height * sizeof(PPMPixel));
    hipMemcpy(cudaBlurData, img->data,
        img->width * img->height * sizeof(PPMPixel),
        hipMemcpyHostToDevice);

    // Even box blur
    for (int i = 0; i < FILTER_SIZE; i++) {
        for (int j = 0; j < FILTER_SIZE; j++) {
            blurKernel[i * FILTER_SIZE + j] = 1.0;
        }
    }
    float *cudaBlurKernel;
    hipMalloc(&cudaBlurKernel, FILTER_SIZE * FILTER_SIZE * sizeof(float));
    hipMemcpy(cudaBlurKernel, blurKernel,
        FILTER_SIZE * FILTER_SIZE * sizeof(float),
        hipMemcpyHostToDevice);

    printf("malloc and cudamalloc and memcpy: %lf\n", currentSeconds() - start);
    start = currentSeconds();
    // Get Walls
    int ltWall = img->width / LTRTWALLDENOM;
    int rtWall = (img->width * (LTRTWALLDENOM - 1)) / LTRTWALLDENOM;
    int tpWall = img->height / TPWALLDENOM;

    // Get color distribution
    range rs[] = {
        {0, ltWall, 0, img->height},
        {rtWall, img->width, 0, img->height},
        {0, img->width, 0, tpWall},
    };

    int i, j, ri;
    for (ri = 0; ri < 3; ri++) {
        range r = rs[ri];
        for (i = r.ymin; i < r.ymax; i++) {
            for (j = r.xmin; j < r.xmax; j++) {
                PPMPixel *pt = getPixel(j, i, img);
                color_counts[
                    getBucketIdx(
                        pt->red / BUCKET_SIZE,
                        pt->green / BUCKET_SIZE,
                        pt->blue / BUCKET_SIZE)
                ] += 1;
            }
        }
    }
    printf("get color_counts: %lf\n", currentSeconds() - start);
    start = currentSeconds();

    int totalBCPix =
        ltWall * img->height +
        (img->width - rtWall) * img->height +
        tpWall * img->width;

    int bcThresh = BCTHRESH_DECIMAL * totalBCPix;

    for (i = 0; i < img->height; i++) {
        for (j = 0; j < img->width; j++) {
            PPMPixel *pt = getPixel(j, i, img);
            unsigned char r = pt->red / BUCKET_SIZE;
            unsigned char g = pt->green / BUCKET_SIZE;
            unsigned char b = pt->blue / BUCKET_SIZE;
            if (color_counts[getBucketIdx(r, g, b)] < bcThresh) {
                oldMask[i * img->width + j] = 1;
            }
        }
    }

    printf("get oldMask: %lf\n", currentSeconds() - start);
    start = currentSeconds();
    memcpy(mask, oldMask, img->width * img->height * sizeof(char));

    // Clean up mask
    for (i = 2; i < img->height - 2; i++) {
        for (j = 2; j < img->width - 2; j++) {
            char thisPx = oldMask[i * img->width + j];
            if (thisPx == 0) {
                int borderSum =
                    oldMask[(i - 1) * img->width + j] +
                    oldMask[i * img->width + j - 1] +
                    oldMask[(i + 1) * img->width + j] +
                    oldMask[i * img->width + j + 1] +
                    oldMask[(i - 2) * img->width + j] +
                    oldMask[i * img->width + j - 2] +
                    oldMask[(i + 2) * img->width + j] +
                    oldMask[i * img->width + j + 2];
                if (borderSum >= 2) {
                    mask[i * img->width + j] = 1;
                }
            }
        }
    }
    printf("get mask: %lf\n", currentSeconds() - start);
    start = currentSeconds();

    // Blur
    printf("finished mask, starting blur\n");
    char *cudaMask;
    hipMalloc(&cudaMask, img->width * img->height * sizeof(char));
    hipMemcpy(cudaMask, mask,
        img->width * img->height * sizeof(char),
        hipMemcpyHostToDevice);

    dim3 threadsPerBlock(SQ_DIM, SQ_DIM);
    dim3 blocks(div_ceil(img->width, SQ_DIM), div_ceil(img->height, SQ_DIM));

    CUDA_CHECK;
    blur<<<blocks, threadsPerBlock>>>(
        img->width,
        img->height,
        cudaImgData,
        cudaBlurKernel,
        cudaBlurData,
        cudaMask
    );

    hipDeviceSynchronize();
    hipMemcpy(
        blurData,
        cudaBlurData,
        img->width * img->height * sizeof(PPMPixel),
        hipMemcpyDeviceToHost
    );

    // Put filter on mask
    int height = img->height;
    int width = img->width;
    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            if (mask[i * width + j] == 1) {
                PPMPixel *pt = getPixel(j, i, img);
                blurData[i * width + j].red = pt->red;
                blurData[i * width + j].green = pt->green;
                blurData[i * width + j].blue = pt->blue;
            }
        }
    }
    printf("get blurData: %lf\n", currentSeconds() - start);
    start = currentSeconds();

    PPMPixel *oldData = img->data;
    img->data = blurData;

    errno = 0;
    writePPM(outfile, img);
    if (errno != 0) {
        exit(EXIT_FAILURE);
    }
    printf("write image: %lf\n", currentSeconds() - start);

    free(oldData);
    free(color_counts);
    free(blurKernel);
    free(img);
    free(img->data);
    hipFree(cudaImgData);
    hipFree(cudaBlurKernel);
    return 0;
}
